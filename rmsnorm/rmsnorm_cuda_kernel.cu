 #include <vector>
#include <iostream>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cublas_helpers.h"


torch::Tensor rmsnorm_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights)
{
  // There is no reason to use more than one stream as every kernel is
  // sequentially dependent
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  const auto batch_size = input.size(0);
  const auto seq_len = input.size(1);
  const auto embed_dim = input.size(2);
  const auto vector_step = batch_size * seq_len;
  const float alpha = 1.0 / embed_dim;
  const float beta = 0.0;

  auto options = torch::TensorOptions().dtype(torch::kFloat32).device(at::kCUDA).requires_grad(false);
  auto channel_variance = torch::zeros({batch_size, seq_len}, options);

  TORCH_CUDABLAS_CHECK_WORKAROUND(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  TORCH_CUDABLAS_CHECK_WORKAROUND(
    hipblasGemmStridedBatchedEx(
      handle,  // handle
      HIPBLAS_OP_T,  // transa
      HIPBLAS_OP_N,  // transb
      1,  // m
      1, // n
      embed_dim, // k
      static_cast<const void *>(&alpha), // alpha
      static_cast<const void *>(input.data_ptr()), // A
      HIP_R_32F, // dtype(A)
      embed_dim,  // lda
      1,          // strideA
      static_cast<const void *>(input.data_ptr()),  // B
      HIP_R_32F, // dtype(B)
      embed_dim, // ldb
      1, // strideB
      static_cast<const void *>(&beta),  // beta
      static_cast<void *>(channel_variance.data_ptr()), // C
      HIP_R_32F, //dtype(C)
      1, // ldc
      1, // strideC
      vector_step, // batchCount
      HIPBLAS_COMPUTE_32F, // computeType
      HIPBLAS_GEMM_DEFAULT) // algo
  );

  return input;
}